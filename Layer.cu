#include "hip/hip_runtime.h"

#include "Layer.h"



__global__ void Sigmid(
    float* __restrict__ dOutputMatOffset,
    const unsigned int subMatSize )
{
    unsigned int eleId = blockDim.x * blockIdx.x + threadIdx.x;
    if (eleId >= subMatSize) return;

    float output = dOutputMatOffset[eleId];
    dOutputMatOffset[eleId] = 1.0f / (1.0f + expf(-output));
}

__global__ void ComputeOutputLayerError(
    float* __restrict__ dErrorMat,
    float* __restrict__ dOutputMat,
    const unsigned short* __restrict__ dClassIndexVec,
    const unsigned int errorMatSize )
{
    unsigned int eleId = blockDim.x * blockIdx.x + threadIdx.x;
    if (eleId >= errorMatSize) return;

    float output = dOutputMat[eleId];
    // For testing
    dOutputMat[eleId] = output;
    dErrorMat[eleId] = output - (float) dClassIndexVec[eleId];
}


Layer::Layer()
{

}

Layer::~Layer()
{
    free( weightMat );
    free( outputMat );
    free( errorMat );
    hipFree( dWeightMat );
    hipFree( dOutputMat );
    hipFree( dErrorMat );
    weightMat = nullptr;
    outputMat = nullptr;
    errorMat = nullptr;
    dWeightMat = nullptr;
    dOutputMat = nullptr;
    dErrorMat = nullptr;
}


void Layer::init(
    const unsigned int numInstances,
    const unsigned int numFeaturesIn,
    const unsigned int numFeaturesOut,
    const unsigned short layerType,
    hipblasHandle_t cublasHandle )
{
    if (layerType == OUTPUT_LAYER && numFeaturesOut == 2)
    {
        printf( "Number of classes in output layer can only be 1"
            "for 2 classes or greater than 2 for more than 2 classes\n" );
        return;
    }

    this->cublasHandle = cublasHandle;
    this->numInstances = numInstances;
    this->numFeaturesOut = numFeaturesOut;
    this->numFeaturesIn = numFeaturesIn;
    this->layerType = layerType;
    numNodes = (layerType == OUTPUT_LAYER) ?
        numFeaturesOut : numFeaturesOut - 1;
    weightMatSize = numFeaturesIn * numNodes;
    errorMatSize = numInstances * numNodes;
    outputMatSize = numInstances * numFeaturesOut;
    inputMatSize = numInstances * numFeaturesIn;

    // Allocate host memo
    weightMat = (float*) malloc( weightMatSize * sizeof( float ) );
    outputMat = (float*) malloc( outputMatSize * sizeof( float ) );
    errorMat = (float*) malloc( errorMatSize * sizeof( float ) );

    // Setup bias in non-output layer
    if (layerType == HIDDEN_LAYER)
    {
        outputOffset = 1;
        // Fill the first feature with X0 for bias
        for (unsigned int i = 0; i < numInstances; i++)
            outputMat[i] = 1;
    }

    // Inie weight matrix
    for (unsigned int i = 0; i < numNodes; i++)
        for (unsigned int j = 0; j < numFeaturesIn; j++)
            // To be randomized
            weightMat[i * numFeaturesIn + j] = 0.0f;

    /* Determine block and grid size of kernel functions */
    if (outputMatSize > 128)
    {
        ccBlockDim.x = 128;
        ccGridDim.x = (outputMatSize + 127) / 128;
    }
    else ccBlockDim.x = outputMatSize;

    if (errorMatSize > 128)
    {
        sigBlockDim.x = 128;
        sigGridDim.x = (errorMatSize + 127) / 128;
    }
    else sigBlockDim.x = errorMatSize;

    // Allocate device memo
    cudaErrorCheck( hipMalloc( (void**) &dWeightMat, weightMatSize * sizeof( float ) ) );
    cudaErrorCheck( hipMalloc( (void**) &dOutputMat, outputMatSize * sizeof( float ) ) );
    cudaErrorCheck( hipMalloc( (void**) &dErrorMat, errorMatSize * sizeof( float ) ) );
    cudaErrorCheck( hipMemcpyAsync(
        dWeightMat,
        weightMat,
        weightMatSize * sizeof( float ),
        hipMemcpyHostToDevice ) );
    // Fill in with X0 as bias
    cudaErrorCheck( hipMemcpyAsync(
        dOutputMat,
        outputMat,
        numInstances * sizeof( float ),
        hipMemcpyHostToDevice ) );

    dOutputMatOffset = (layerType != HIDDEN_LAYER) ? dOutputMat : dOutputMat + numInstances;
}

float* Layer::forwardOutput( const float* dInputMat )
{
    // use cublasCgemm3m ...


    const float alpha = 1.0f;
    const float beta = 0.0f;

    cublasErrorCheck( hipblasSgemm(
        cublasHandle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        numInstances,
        numNodes,
        numFeaturesIn,
        &alpha,
        dInputMat,
        numInstances,
        dWeightMat,
        numFeaturesIn,
        &beta,
        dOutputMatOffset,
        numInstances ) );
    Sigmid<<< sigGridDim, sigBlockDim >>>(
        dOutputMatOffset,
        // Error mat size = output mat size without X0s
        errorMatSize );
    cudaErrorCheck( hipGetLastError() );

    return dOutputMat;
}

void Layer::backPropError(
    float* preLayerErrorMat,
    const float* inputMat )
{
    unsigned int numNodesPreLayer = numFeaturesIn - 1;
    // Ignore bias input
    unsigned int offset = 1;
    for (unsigned int i = 0; i < numInstances; i++)
        for (unsigned int idIn = 0; idIn < numNodesPreLayer; idIn++)
        {
            float sum = 0.0f;
            for (unsigned int idNode = 0; idNode < numNodes; idNode++)
                sum += weightMat[idNode * numFeaturesIn + idIn + offset] *
                    errorMat[numNodes * i + idNode];
            preLayerErrorMat[numNodesPreLayer * i + idIn] =
                sum * inputMat[numFeaturesIn * i + idIn + offset] *
                (1.0f - inputMat[numFeaturesIn * i + idIn + offset]);
        }
}

void Layer::updateWeights(
    const float* inputMat,
    const float learningRate )
{
    for (unsigned int idNode = 0; idNode < numNodes; idNode++)
        for (unsigned int idIn = 0; idIn < numFeaturesIn; idIn++)
        {
            float sum = 0.0f;
            for (unsigned int i = 0; i < numInstances; i++)
                sum += inputMat[numFeaturesIn * i + idIn] *
                    errorMat[numNodes * i + idNode];
            weightMat[numFeaturesIn * idNode + idIn] -=
                learningRate / (float) numInstances * sum;
        }

    printf( "Back propagate completed, weight: %f\n", weightMat[0] );
}

void Layer::computeOutputLayerError(
    const unsigned short* dClassIndexVec,
    const unsigned short* classIndexVec )
{
    if (layerType != OUTPUT_LAYER)
    {
        printf( "computeOutputLayerError() can only be ran by output layer.\n" );
        return;
    }

    ComputeOutputLayerError<<< ccGridDim, ccBlockDim >>>(
        dErrorMat,
        dOutputMat,
        dClassIndexVec,
        errorMatSize );
    cudaErrorCheck( hipGetLastError() );

    // Copy from device to host
    // For testing gradient descent
    cudaErrorCheck( hipMemcpy(
        outputMat,
        dOutputMat,
        outputMatSize * sizeof( float ),
        hipMemcpyDeviceToHost ) );

    float costSum = 0.0f;
    for (unsigned int i = 0; i < numInstances; i++)
        for (unsigned int j = 0; j < numNodes; j++)
            costSum -= (classIndexVec[i]) ?
                logf(outputMat[i * numNodes + j]) : logf(1.0f - outputMat[i * numNodes + j]);

    printf( "Cost: %f\n", costSum );
}

float* Layer::getDWeightPtr()
{
    return dWeightMat;
}

float* Layer::getDOutputPtr()
{
    return dOutputMat;
}

float* Layer::getDErrorPtr()
{
    return dErrorMat;
}

float* Layer::getWeightPtr()
{
    return weightMat;
}

float* Layer::getOutputPtr()
{
    return outputMat;
}

float* Layer::getErrorPtr()
{
    return errorMat;
}
